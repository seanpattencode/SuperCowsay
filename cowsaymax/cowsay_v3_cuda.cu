#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DEFAULT_WIDTH 40
#define THREADS_PER_BLOCK 256

__global__ void fill_pattern_kernel(char *buffer, char pattern, int count, int offset) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count) {
        buffer[offset + idx] = pattern;
    }
}

__global__ void copy_message_kernel(char *buffer, const char *msg, int len, int offset) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        buffer[offset + idx] = msg[idx];
    }
}

void cowsay_cuda(const char *msg, int width) {
    int len = strlen(msg);
    int w = len <= width ? len : width;
    int buffer_size = 8192;
    
    char *h_buffer = (char*)malloc(buffer_size);
    char *d_buffer;
    char *d_msg;
    
    hipMalloc(&d_buffer, buffer_size);
    hipMalloc(&d_msg, len + 1);
    hipMemcpy(d_msg, msg, len + 1, hipMemcpyHostToDevice);
    
    char *ptr = h_buffer;
    int offset = 0;
    
    *ptr++ = ' ';
    offset++;
    
    int blocks = (w + 2 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    fill_pattern_kernel<<<blocks, THREADS_PER_BLOCK>>>(d_buffer, '_', w + 2, offset);
    hipDeviceSynchronize();
    offset += w + 2;
    ptr += w + 2;
    
    hipMemcpy(h_buffer + 1, d_buffer + 1, w + 2, hipMemcpyDeviceToHost);
    *ptr++ = '\n';
    offset++;
    
    if (len <= width) {
        strcpy(ptr, "< ");
        ptr += 2;
        offset += 2;
        
        blocks = (len + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        copy_message_kernel<<<blocks, THREADS_PER_BLOCK>>>(d_buffer, d_msg, len, offset);
        hipDeviceSynchronize();
        hipMemcpy(ptr, d_buffer + offset, len, hipMemcpyDeviceToHost);
        
        ptr += len;
        offset += len;
        strcpy(ptr, " >\n");
        ptr += 3;
        offset += 3;
    } else {
        for (int i = 0; i < len; i += width) {
            int chunk = (len - i < width) ? len - i : width;
            char c1 = i == 0 ? '/' : (i + width >= len ? '\\' : '|');
            char c2 = i == 0 ? '\\' : (i + width >= len ? '/' : '|');
            
            *ptr++ = c1;
            *ptr++ = ' ';
            offset += 2;
            
            blocks = (chunk + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
            copy_message_kernel<<<blocks, THREADS_PER_BLOCK>>>(d_buffer, d_msg + i, chunk, offset);
            hipDeviceSynchronize();
            hipMemcpy(ptr, d_buffer + offset, chunk, hipMemcpyDeviceToHost);
            
            ptr += chunk;
            offset += chunk;
            
            int padding = width - chunk;
            if (padding > 0) {
                blocks = (padding + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
                fill_pattern_kernel<<<blocks, THREADS_PER_BLOCK>>>(d_buffer, ' ', padding, offset);
                hipDeviceSynchronize();
                hipMemcpy(ptr, d_buffer + offset, padding, hipMemcpyDeviceToHost);
                ptr += padding;
                offset += padding;
            }
            
            *ptr++ = ' ';
            *ptr++ = c2;
            *ptr++ = '\n';
            offset += 3;
        }
    }
    
    *ptr++ = ' ';
    offset++;
    
    blocks = (w + 2 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    fill_pattern_kernel<<<blocks, THREADS_PER_BLOCK>>>(d_buffer, '-', w + 2, offset);
    hipDeviceSynchronize();
    hipMemcpy(ptr, d_buffer + offset, w + 2, hipMemcpyDeviceToHost);
    ptr += w + 2;
    
    const char *cow = "\n        \\   ^__^\n         \\  (oo)\\_______\n            (__)\\       )\\/\\\n                ||----w |\n                ||     ||\n";
    strcpy(ptr, cow);
    ptr += strlen(cow);
    
    fwrite(h_buffer, 1, ptr - h_buffer, stdout);
    
    hipFree(d_buffer);
    hipFree(d_msg);
    free(h_buffer);
}

int main(int argc, char *argv[]) {
    char msg[1024] = "Hello, World!";
    int width = DEFAULT_WIDTH;
    size_t remaining = sizeof(msg) - 1;
    
    int has_message = 0;
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "-w") == 0 && i + 1 < argc) {
            width = atoi(argv[++i]);
        } else if (strcmp(argv[i], "--stdin") == 0) {
            fgets(msg, sizeof(msg), stdin);
            msg[strcspn(msg, "\n")] = 0;
            has_message = 1;
        } else {
            if (!has_message) {
                msg[0] = '\0';
                remaining = sizeof(msg) - 1;
                has_message = 1;
            }
            size_t current_len = strlen(msg);
            size_t arg_len = strlen(argv[i]);
            size_t space_needed = (current_len > 0) ? 1 : 0;
            
            if (arg_len + space_needed > remaining) break;
            
            if (current_len > 0) {
                strcat(msg, " ");
                remaining--;
            }
            strncat(msg, argv[i], remaining);
            remaining -= arg_len;
        }
    }
    
    cowsay_cuda(msg, width);
    return 0;
}